#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include "debug.cuh"
#include <memory>
#include <fstream>
#include <random>
const int warpSize = 32;
const int BLOCK_SIZE = 1024; // 设置固定的block大小为1024

__device__ double atomicAddDouble(double *address, double val)
{
    unsigned long long int *address_as_ull = (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void dotProductKernel(double *x, double *y, int n, double *partialSums)
{
    unsigned FULL_MASK = 0xffffffff;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = threadIdx.x % warpSize;
    __shared__ double warpPartialSum[warpSize]; // 每个warp一个shared memory位置

    double sum = 0;
    if (index < n)
    {
        sum = x[index] * y[index];
    }
    else
    {
        sum = 0;
    }

    // Warp内部求和
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
    }

    // 每个warp的第一个线程将结果写入共享内存
    if (lane == 0)
    {
        warpPartialSum[threadIdx.x / warpSize] = sum;
    }
    __syncthreads();

    // block内的第一个warp将所有warp的结果汇总
    if (threadIdx.x < warpSize)
    {
        sum = (threadIdx.x < blockDim.x / warpSize) ? warpPartialSum[lane] : 0;
        for (int offset = warpSize / 2; offset > 0; offset /= 2)
        {
            sum += __shfl_down_sync(FULL_MASK, sum, offset);
        }
    }

    // 将block的结果写入全局内存
    if (threadIdx.x == 0)
    {
        partialSums[blockIdx.x] = sum;
    }
}

__global__ void MatrixVectorProduct(double *matrix, double *vector, double *result, int rows, int cols)
{
    int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = globalThreadIdx / warpSize;    // 计算全局warp ID
    int warpCount = (blockDim.x) / warpSize;    // 计算单个warp数量
    int rowsPerWarp = max(1, rows / warpCount); // 每个warp处理的行数
    // __shared__ double shared_vector[BLOCK_SIZE];
    // for (int i = 0; i < cols; i += warpSize)
    // {
    //     if (threadIdx.x + i < cols)
    //     {
    //         shared_vector[threadIdx.x + i] = vector[threadIdx.x + i];
    //     }
    // }
    __syncthreads();
    int extraRows = rows % warpCount; // 不能均匀分配的额外行数

    int startRow = warpId * rowsPerWarp + min(warpId, extraRows);
    int endRow = startRow + rowsPerWarp + (warpId < extraRows ? 1 : 0);

    for (int row = startRow; row < endRow; row++)
    {
        double sum = 0;
        for (int col = threadIdx.x % warpSize; col < cols; col += warpSize)
        {
            sum += matrix[row * cols + col] * vector[col];
        }
        for (int offset = warpSize / 2; offset > 0; offset /= 2)
        {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (threadIdx.x % warpSize == 0)
        {
            atomicAddDouble(&result[row], sum);
        }
    }
}

void init_matrix(std::vector<double> &matrix, int row, int col)
{
    matrix.clear();
    std::random_device rd;  // 随机数种子
    std::mt19937 gen(rd()); // 使用Mersenne Twister算法的生成器
                            // 定义[0, 1)范围的均匀分布
    std::uniform_real_distribution<> distrib(0.0, 0.1);
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {

            matrix.push_back(distrib(gen));

            // if (i == j)
            // {
            //     matrix.push_back(1);
            // }
            // else
            // {
            //     matrix.push_back(0);
            // }
        }
    }
}
void init_vector(std::vector<double> &vector, int row)
{
    vector.clear();
    std::random_device rd;  // 随机数种子
    std::mt19937 gen(rd()); // 使用Mersenne Twister算法的生成器

    // 定义[0, 1)范围的均匀分布
    std::uniform_real_distribution<> distrib(0.0, 0.01);

    for (int i = 0; i < row; i++)
    {
        vector.push_back(distrib(gen));
        // vector.push_back(1);
    }
}
void init_result(std::vector<double> &result, int row)
{
    result.clear();
    for (int i = 0; i < row; i++)
    {
        result.push_back(0);
    }
}
void initCudaMemory(double **d_ptr, const std::vector<double> &host_data)
{
    size_t size = host_data.size() * sizeof(double);
    CHECK(hipMalloc(d_ptr, size));                                            // 在GPU上分配内存
    CHECK(hipMemcpy(*d_ptr, host_data.data(), size, hipMemcpyHostToDevice)); // 复制数据到GPU
}

void processMatrixInStreams(const std::vector<double> &matrix,
                            const std::vector<double> &vector,
                            std::vector<double> &result,
                            int M, int numRows, int numCols, float &cputime, float &gputime)
{
    // 确保矩阵的大小适合分块
    int rowsPerBlock = numRows / M;  // 每个块的行数
    int remainingRows = numRows % M; // 剩余的行数

    // 创建并初始化CUDA流
    hipStream_t streams[M];
    for (int i = 0; i < M; ++i)
    {
        hipStreamCreate(&streams[i]);
    }

    // 分配设备内存
    double *d_matrix, *d_result, *d_vector;
    hipMalloc(&d_matrix, matrix.size() * sizeof(double));
    hipMalloc(&d_vector, vector.size() * sizeof(double));
    hipMalloc(&d_result, result.size() * sizeof(double));

    // 复制数据到设备
    hipMemcpy(d_vector, vector.data(), vector.size() * sizeof(double), hipMemcpyHostToDevice);
    auto cpu_start = std::chrono::high_resolution_clock::now();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start);

    // 在每个流上启动核函数
    for (int i = 0; i < M; ++i)
    {
        int startRow = i * rowsPerBlock;
        int rowsToProcess = (i == M - 1) ? rowsPerBlock + remainingRows : rowsPerBlock; // 最后一个块处理额外的行数
        hipMemcpyAsync(d_matrix + startRow * numCols,
                        matrix.data() + startRow * numCols,
                        rowsToProcess * numCols * sizeof(double),
                        hipMemcpyHostToDevice, streams[i]);
        MatrixVectorProduct<<<1, BLOCK_SIZE, 0, streams[i]>>>(d_matrix + startRow * numCols,
                                                              d_vector, d_result + startRow, rowsToProcess, numCols);
        hipMemcpyAsync(result.data() + startRow,
                        d_result + startRow,
                        rowsToProcess * sizeof(double),
                        hipMemcpyDeviceToHost, streams[i]);
    }

    // 等待所有流完成
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    hipEventRecord(stop);
    std::chrono::duration<double> duration = end - cpu_start;
    cputime = duration.count() * 1000;
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    gputime = milliseconds;
    // 释放资源
    for (int i = 0; i < M; ++i)
    {
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_result);
}

int main()
{
    std::cout << "Program started." << std::endl;
    // 创建随机数生成器
    std::random_device rd;
    std::mt19937 gen(rd());

    // 定义范围为1000到2000的均匀分布
    std::uniform_int_distribution<> distrib(1000, 2048);
    // 打开CSV文件用于写入
    std::ofstream outFile("results.csv");
    // 写入标题行
    if (outFile.is_open())
    {
        outFile << "M,CPU-Time(ms),GPU-Time(ms)\n";
    }

    const int numTrials = 1000; // 设置重复执行的次数

    std::vector<double> matrix, vector, result;

    for (int M = 1; M <= 16; M++)
    {
        double totalGPUTime = 0;
        double totalCPUTime = 0;
        for (int trial = 0; trial < numTrials; ++trial)
        {
            int N = distrib(gen);
            int K = distrib(gen);
            init_matrix(matrix, N, K);
            init_vector(vector, K);
            init_result(result, N);
            float gputime = 0;
            float cputime = 0;
            processMatrixInStreams(matrix, vector, result, M, N, K, cputime, gputime);
            totalGPUTime += gputime;
            totalCPUTime += cputime;
        }
        float sum = 0;
        for (const auto element : result)
        {
            sum += element;
        }

        std::cout << "M: " << M << "\t"
                  << "CPU-time: " << totalCPUTime << "\t"
                  << "GPU-time: " << totalGPUTime << std::endl;

        std::cout << "sum: " << sum << std::endl;
        // 将M和time写入到文件
        outFile << M << "," << totalCPUTime << "," << totalGPUTime << "\n";
    }

    // 关闭文件
    outFile.close();

    std::cout << "Program ended." << std::endl;
    return 0;
}
